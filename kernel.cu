
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE  16        // submatrix size
#define N           1024       // matrix size is N*N
hipError_t addWithCuda(int *a, int *b, int *c, int size);

__global__ void addKernel(int *a, int *b, int *c)
{
		int bx = blockIdx.x;
		int by = blockIdx.y;
		int tx = threadIdx.x;
		int ty = threadIdx.y;
		int sum = 0;            // computed subelement
		int   ia = N * BLOCK_SIZE * by + N * ty;   // a [i][0]
		int   ib = BLOCK_SIZE * bx + tx;

		for (int k = 0; k < N; k++)
			sum += a[ia + k] * b[ib + k*N];
}

int main()
{
	int Sizebytes = N*N*sizeof(int);
	int *a, *b, *c;

	hipError_t cudaStatus;
	a = (int*)malloc(N*N*sizeof(int));
	b = (int*)malloc(N*N*sizeof(int));
	c = (int*)malloc(N*N*sizeof(int));

	for (int i = 0; i < N; i++)
		for (int j = 0; j < N; j++)
		{
			int	k = N*i + j;

			a[k] = rand() % 100 + 1;
			b[k] = rand() % 100 + 1;
		}
	for (int i = 0; i < 20; i++){
		cudaStatus = addWithCuda(a, b, c, Sizebytes);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "addWithCuda failed!");
			return 1;
		}
		cudaStatus = hipDeviceReset();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceReset failed!");
			return 1;
		}
	}
	free(a);
	free(b);
	free(c);
    return 0;
}

//������� ��� ���������� �������� � gpu 
hipError_t addWithCuda(int *a, int *b, int *c, int size)
{
	int *dev_a, *dev_b, *dev_c;
    hipError_t cudaStatus;
	dim3 threads(16, 16);
	dim3 blocks(N / threads.x, N / threads.y);

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<blocks, threads>>>(dev_a, dev_b, dev_c);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
